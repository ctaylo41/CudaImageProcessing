#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include "imageTools.h"
#include "stb_image_write.h"
#include <vector>
inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess)
  {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

__global__ void imageLoad(unsigned char *image, uchar4 *imageLoaded, size_t imgSize)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = index; i < imgSize; i += stride)
  {
    imageLoaded[i].x = image[i * 4 + 0];
    imageLoaded[i].y = image[i * 4 + 1];
    imageLoaded[i].z = image[i * 4 + 2];
    imageLoaded[i].w = image[i * 4 + 3];
  }
}

__global__ void imageWrite(unsigned char *image, uchar4 *pixels, int width, int height)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < width * height; i += stride)
  {
    image[i * 4 + 0] = pixels[i].x;
    image[i * 4 + 1] = pixels[i].y;
    image[i * 4 + 2] = pixels[i].z;
    image[i * 4 + 3] = pixels[i].w;
  }
}

void imageWriteWrapper(const char *filename, uchar4 *pixels, int width, int height)
{
  unsigned char *image;
  uchar4 *d_pixels;

  checkCuda(hipMallocManaged(&image, width * height * 4 * sizeof(unsigned char)));
  checkCuda(hipMallocManaged(&d_pixels, width * height * sizeof(uchar4)));
  checkCuda(hipMemcpy(d_pixels, pixels, width * height * sizeof(uchar4), hipMemcpyHostToDevice));

  int threadsPerBlock = 256;
  int numBlocks = (width * height + threadsPerBlock - 1) / threadsPerBlock;

  imageWrite<<<numBlocks, threadsPerBlock>>>(image, d_pixels, width, height);
  checkCuda(hipGetLastError());
  hipDeviceSynchronize();
  unsigned char *d_image = (unsigned char *)malloc(width * height * 4 * sizeof(unsigned char));
  hipMemcpy(d_image, image, width * height * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost);
  stbi_write_png(filename, width, height, 4, d_image, width * 4);

  hipFree(d_pixels);
  hipFree(image);
}

void imageLoadWrapper(unsigned char *image, uchar4 *imageLoaded, size_t imgSize)
{
  unsigned char *d_image;
  uchar4 *d_imageLoaded;

  checkCuda(hipMallocManaged(&d_image, imgSize * 4 * sizeof(unsigned char)));
  checkCuda(hipMallocManaged(&d_imageLoaded, imgSize * sizeof(uchar4)));
  checkCuda(hipMemcpy(d_image, image, imgSize * 4 * sizeof(unsigned char), hipMemcpyHostToDevice));

  int imgSizeInt = (int)imgSize;
  int threadsPerBlock = 256;
  int numBlocks = (imgSizeInt + threadsPerBlock - 1) / threadsPerBlock;
  imageLoad<<<numBlocks, threadsPerBlock>>>(d_image, d_imageLoaded, imgSize);
  checkCuda(hipGetLastError());
  hipDeviceSynchronize();
  hipMemcpy(imageLoaded, d_imageLoaded, imgSize * sizeof(uchar4), hipMemcpyDeviceToHost);
  hipFree(d_image);
  hipFree(d_imageLoaded);
}

__global__ void imageGrayScale(uchar4 *returnImage, uchar4 *imageLoaded, size_t imgSize)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = index; i < imgSize; i += stride)
  {
    uchar4 pixel = imageLoaded[i];
    unsigned char gray = (unsigned char)(0.299f * pixel.x + 0.587f * pixel.y + 0.114f * pixel.z);
    returnImage[i].x = gray;
    returnImage[i].y = gray;
    returnImage[i].z = gray;
    returnImage[i].w = pixel.w;
  }
}

void imageGrayScaleWrapper(uchar4 *returnImage, uchar4 *imageLoaded, size_t imgSize)
{
  uchar4 *d_returnImage;
  uchar4 *d_imageLoaded;

  checkCuda(hipMallocManaged(&d_returnImage, imgSize * sizeof(uchar4)));
  checkCuda(hipMallocManaged(&d_imageLoaded, imgSize * sizeof(uchar4)));
  checkCuda(hipMemcpy(d_imageLoaded, imageLoaded, imgSize * sizeof(uchar4), hipMemcpyHostToDevice));

  int imgSizeInt = (int)imgSize;
  int threadsPerBlock = 256;
  int numBlocks = (imgSizeInt + threadsPerBlock - 1) / threadsPerBlock;
  imageGrayScale<<<numBlocks, threadsPerBlock>>>(d_returnImage, d_imageLoaded, imgSize);
  checkCuda(hipGetLastError());
  hipDeviceSynchronize();
  hipMemcpy(returnImage, d_returnImage, imgSize * sizeof(uchar4), hipMemcpyDeviceToHost);
  hipFree(d_returnImage);
  hipFree(d_imageLoaded);
}

__global__ void imageSobelEdge(uchar4 *returnImage, uchar4 *imageLoaded, int width, int height)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x > 0 && y > 0 && x < width - 1 && y < height - 1)
  {
    float Gx = 0;
    float Gy = 0;

    int sobelX[3][3] = {{-1, 0, 1}, {-2, 0, 2}, {-1, 0, 1}};
    int sobelY[3][3] = {{-1, -2, -1}, {0, 0, 0}, {1, 2, 1}};

    for (int ky = -1; ky <= 1; ky++)
    {
      for (int kx = -1; kx <= 1; kx++)
      {
        int idx = (y + ky) * width + (x + kx);
        uchar4 pixel = imageLoaded[idx];
        Gx += pixel.x * sobelX[ky + 1][kx + 1];
        Gy += pixel.x * sobelY[ky + 1][kx + 1];
      }
    }
    float magnitude = sqrt(Gx * Gx + Gy * Gy);
    magnitude = max(0.0f, min(255.0f, magnitude));
    returnImage[y * width + x].x = magnitude;
    returnImage[y * width + x].y = magnitude;
    returnImage[y * width + x].z = magnitude;
    returnImage[y * width + x].w = 255;
  }
}

__global__ void imageGaussianBlur(uchar4 *returnImage, uchar4 *imageLoaded, int width, int height, int kernalSize, float *kernal)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x > 0 && y > 0 && x < width - 1 && y < height - 1)
  {
    float3 sum = make_float3(0.0f, 0.0f, 0.0f);
    for (int ky = -kernalSize / 2; ky <= kernalSize / 2; ky++)
    {
      for (int kx = -kernalSize / 2; kx <= kernalSize / 2; kx++)
      {
        int idx = (y + ky) * width + (x + kx);
        uchar4 pixel = imageLoaded[idx];
        float kernalValue = kernal[(ky + kernalSize / 2) * kernalSize + (kx + kernalSize / 2)];
        sum.x += pixel.x * kernalValue;
        sum.y += pixel.y * kernalValue;
        sum.z += pixel.z * kernalValue;
      }
    }
    returnImage[y * width + x].x = sum.x;
    returnImage[y * width + x].y = sum.y;
    returnImage[y * width + x].z = sum.z;
    returnImage[y * width + x].w = 255;
  }
}

__global__ void imageMeanBlur(uchar4 *returnImage, uchar4 *imageLoaded, int width, int height, int kernalSize)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x > 0 && y > 0 && x < width - 1 && y < height - 1)
  {
    int3 mean = make_int3(0, 0, 0);
    for (int ky = -kernalSize / 2; ky <= kernalSize / 2; ky++)
    {
      for (int kx = -kernalSize / 2; kx <= kernalSize / 2; kx++)
      {
        int idx = (y + ky) * width + (x + kx);
        uchar4 pixel = imageLoaded[idx];
        mean.x += pixel.x;
        mean.y += pixel.y;
        mean.z += pixel.z;
      }
    }
    
    mean.x = (mean.x/(kernalSize*kernalSize));
    mean.y = (mean.y/(kernalSize*kernalSize));
    mean.z = (mean.z/(kernalSize*kernalSize));
    returnImage[y * width + x].x = mean.x;
    returnImage[y * width + x].y = mean.y;
    returnImage[y * width + x].z = mean.z;
    returnImage[y * width + x].w = 255;
  }
}

void imageSobelEdgeWrapper(uchar4 *returnImage, uchar4 *imageLoaded, int width, int height)
{
  uchar4 *d_returnImage;
  uchar4 *d_imageLoaded;
  checkCuda(hipMallocManaged(&d_returnImage, width * height * sizeof(uchar4)));
  checkCuda(hipMallocManaged(&d_imageLoaded, width * height * sizeof(uchar4)));
  checkCuda(hipMemcpy(d_imageLoaded, imageLoaded, width * height * sizeof(uchar4), hipMemcpyHostToDevice));
  int threadsPerBlock = 16;
  int numBlocksX = (width + threadsPerBlock - 1) / threadsPerBlock;
  int numBlocksY = (height + threadsPerBlock - 1) / threadsPerBlock;
  dim3 blocks(numBlocksX, numBlocksY);
  dim3 threads(threadsPerBlock, threadsPerBlock);
  imageSobelEdge<<<blocks, threads>>>(d_returnImage, d_imageLoaded, width, height);
  checkCuda(hipGetLastError());
  checkCuda(hipDeviceSynchronize());
  hipMemcpy(returnImage, d_returnImage, width * height * sizeof(uchar4), hipMemcpyDeviceToHost);
  hipFree(d_returnImage);
  hipFree(d_imageLoaded);
}

float *generateGaussianKernal(int size, float sigma)
{
  float *kernal = (float *)malloc(size * size * sizeof(float));
  float sum = 0.0f;
  int halfSize = size / 2;
  for (int i = -halfSize; i <= halfSize; i++)
  {
    for (int j = -halfSize; j <= halfSize; j++)
    {
      float value = exp(-(i * i + j * j) / (2 * sigma * sigma)) / (2 * M_PI * sigma * sigma);
      kernal[(i + halfSize) * size + (j + halfSize)] = value;
      sum += value;
    }
  }

  for (int i = 0; i < size; i++)
  {
    for (int j = 0; j < size; j++)
    {
      kernal[i * size + j] /= sum;
    }
  }
  return kernal;
}

void imageGaussianBlurWrapper(uchar4 *returnImage, uchar4 *imageLoaded, int width, int height, int size, float sigma)
{
  float *kernal = generateGaussianKernal(size, sigma);
  uchar4 *d_returnImage;
  uchar4 *d_imageLoaded;
  float *d_kernal;
  checkCuda(hipMallocManaged(&d_returnImage, width * height * sizeof(uchar4)));
  checkCuda(hipMallocManaged(&d_imageLoaded, width * height * sizeof(uchar4)));
  checkCuda(hipMallocManaged(&d_kernal, size * size * sizeof(float)));
  checkCuda(hipMemcpy(d_imageLoaded, imageLoaded, width * height * sizeof(uchar4), hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(d_kernal, kernal, size * size * sizeof(float), hipMemcpyHostToDevice));
  int threadsPerBlock = 16;
  int numBlocksX = (width + threadsPerBlock - 1) / threadsPerBlock;
  int numBlocksY = (height + threadsPerBlock - 1) / threadsPerBlock;
  dim3 blocks(numBlocksX, numBlocksY);
  dim3 threads(threadsPerBlock, threadsPerBlock);
  imageGaussianBlur<<<blocks, threads>>>(d_returnImage, d_imageLoaded, width, height, size, d_kernal);
  checkCuda(hipGetLastError());
  hipDeviceSynchronize();
  hipMemcpy(returnImage, d_returnImage, width * height * sizeof(uchar4), hipMemcpyDeviceToHost);
  hipFree(d_returnImage);
  hipFree(d_imageLoaded);
  hipFree(d_kernal);
  free(kernal);
}

void imageMeanBlurWrapper(uchar4 *returnImage, uchar4 *imageLoaded, int width, int height)
{
  uchar4 *d_image_return;
  uchar4 *d_image_loaded;
  checkCuda(hipMalloc(&d_image_loaded, width * height * sizeof(uchar4)));
  checkCuda(hipMalloc(&d_image_return, width * height * sizeof(uchar4)));
  checkCuda(hipMemcpy(d_image_loaded, imageLoaded, width * height * sizeof(uchar4), hipMemcpyHostToDevice));
  int threadsPerBlock = 16;
  int numBlocksX = (width + threadsPerBlock - 1) / threadsPerBlock;
  int numBlocksY = (height + threadsPerBlock - 1) / threadsPerBlock;
  dim3 blocks(numBlocksX, numBlocksY);
  dim3 threads(threadsPerBlock, threadsPerBlock);
  imageMeanBlur<<<blocks, threads>>>(d_image_return, d_image_loaded, width, height, 3);
  checkCuda(hipGetLastError());
  hipDeviceSynchronize();
  hipMemcpy(returnImage, d_image_return, width * height * sizeof(uchar4), hipMemcpyDeviceToHost);
  hipFree(d_image_return);
  hipFree(d_image_loaded);
}
